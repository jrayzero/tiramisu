//
// Created by Jessica Ray on 12/6/17.
//

#include <cassert>
#include "tiramisu/tiramisu_cuda.h"
#include <stdio.h>

#ifdef DRIVER
extern "C" {

  /*void tiramisu_cuda_malloc(hipDeviceptr_t *device_ptr, size_t bytes) {
  assert(hipMalloc(device_ptr, bytes) == 0 && "tiramisu_cuda_malloc failed");
}

void tiramisu_cuda_free(hipDeviceptr_t device_ptr) {
    assert(hipFree(device_ptr) == 0 && "tiramisu_cuda_free failed");
}

  void tiramisu_cuda_memcpy_h2d(hipDeviceptr_t dst, const void *src, size_t count) {
    assert(hipMemcpyHtoD(dst, src, count) == 0 && "tiramisu_cuda_memcpy_h2d failed");
  }

  void tiramisu_cuda_memcpy_d2h(void *dst, hipDeviceptr_t src, size_t count) {
    assert(hipMemcpyDtoH(dst, src, count) == 0 && "tiramisu_cuda_memcpy_d2h failed");
  }

  void htiramisu_cuda_malloc(halide_buffer_t *buff, size_t bytes) {
    hipDeviceptr_t p;
    tiramisu_cuda_malloc(&p, bytes);
    buff->device = p;
  }

  void htiramisu_cuda_free(halide_buffer_t *buff) {
    tiramisu_cuda_free((hipDeviceptr_t)(buff->device));
  }

  void htiramisu_cuda_memcpy_h2d(halide_buffer_t *dst, const void *src, size_t count) {
    tiramisu_cuda_memcpy_h2d((hipDeviceptr_t)(dst->device), src, count);
  }

  void htiramisu_cuda_memcpy_d2h(void *dst, halide_buffer_t *src, size_t count) {
    tiramisu_cuda_memcpy_d2h(dst, (hipDeviceptr_t)(src->device), count);
  }*/

}

#endif

#ifdef RUNTIME

extern "C" {

void tiramisu_cuda_malloc(void **device_ptr, size_t bytes) {
    assert(hipMalloc(device_ptr, bytes) == 0 && "tiramisu_cuda_malloc failed");
}

void htiramisu_cuda_memcpy_h2d(halide_buffer_t *dst, const void *src, size_t count, size_t dst_offset) {
    tiramisu_cuda_memcpy_h2d(&(((float*)(dst->device))[dst_offset]), src, count);
}

void htiramisu_cuda_memcpy_d2h(void *dst, halide_buffer_t *src, size_t count, size_t src_offset) {
    tiramisu_cuda_memcpy_d2h(dst, &(((float*)(src->device))[src_offset]), count);
}

void tiramisu_cuda_free(void *device_ptr) {
    assert(hipFree(device_ptr) == 0 && "tiramisu_cuda_free failed");
}

void tiramisu_cuda_memcpy_h2d(void *dst, const void *src, size_t count) {
    assert(hipMemcpy(dst, src, count, hipMemcpyHostToDevice) == 0 && "tiramisu_cuda_memcpy_h2d failed");
}

void tiramisu_cuda_memcpy_h2h(void *dst, const void *src, size_t count) {
    assert(hipMemcpy(dst, src, count, hipMemcpyHostToHost) == 0 && "tiramisu_cuda_memcpy_h2h failed");
}

void tiramisu_cuda_memcpy_d2h(void *dst, const void *src, size_t count) {
    assert(hipMemcpy(dst, src, count, hipMemcpyDeviceToHost) == 0 && "tiramisu_cuda_memcpy_d2h failed");
}

void tiramisu_cuda_memcpy_d2d(void *dst, const void *src, size_t count) {
    assert(hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice) == 0 && "tiramisu_cuda_memcpy_d2d failed");
}

void tiramisu_cuda_memcpy_h2d_async(void *dst, const void *src, size_t count, hipStream_t stream) {
    assert(false && "Not implemented yet");
    assert(hipMemcpyAsync(dst, src, count, hipMemcpyHostToDevice, stream) == 0 && "tiramisu_cuda_memcpy_h2d_async failed");
}

void tiramisu_cuda_memcpy_h2h_async(void *dst, const void *src, size_t count, hipStream_t stream) {
    assert(false && "Not implemented yet");
    assert(hipMemcpyAsync(dst, src, count, hipMemcpyHostToHost, stream) == 0 && "tiramisu_cuda_memcpy_h2h_async failed");
}

void tiramisu_cuda_memcpy_d2h_async(void *dst, const void *src, size_t count, hipStream_t stream) {
    assert(false && "Not implemented yet");
    assert(hipMemcpyAsync(dst, src, count, hipMemcpyDeviceToHost, stream) == 0 && "tiramisu_cuda_memcpy_d2h_async failed");
}

void tiramisu_cuda_memcpy_d2d_async(void *dst, const void *src, size_t count, hipStream_t stream) {
    assert(false && "Not implemented yet");
    assert(hipMemcpyAsync(dst, src, count, hipMemcpyDeviceToDevice, stream) == 0 && "tiramisu_cuda_memcpy_d2d_async failed");
}

}
#endif