//
// Created by Jessica Ray on 12/6/17.
//

#include <cassert>
#include "tiramisu/tiramisu_cuda.h"
#include <stdio.h>

#ifdef DRIVER
extern "C" {

  /*void tiramisu_cuda_malloc(hipDeviceptr_t *device_ptr, size_t bytes) {
  assert(hipMalloc(device_ptr, bytes) == 0 && "tiramisu_cuda_malloc failed");
}

void tiramisu_cuda_free(hipDeviceptr_t device_ptr) {
    assert(hipFree(device_ptr) == 0 && "tiramisu_cuda_free failed");
}

  void tiramisu_cuda_memcpy_h2d(hipDeviceptr_t dst, const void *src, size_t count) {
    assert(hipMemcpyHtoD(dst, src, count) == 0 && "tiramisu_cuda_memcpy_h2d failed");
  }

  void tiramisu_cuda_memcpy_d2h(void *dst, hipDeviceptr_t src, size_t count) {
    assert(hipMemcpyDtoH(dst, src, count) == 0 && "tiramisu_cuda_memcpy_d2h failed");
  }

  void htiramisu_cuda_malloc(halide_buffer_t *buff, size_t bytes) {
    hipDeviceptr_t p;
    tiramisu_cuda_malloc(&p, bytes);
    buff->device = p;
  }

  void htiramisu_cuda_free(halide_buffer_t *buff) {
    tiramisu_cuda_free((hipDeviceptr_t)(buff->device));
  }

  void htiramisu_cuda_memcpy_h2d(halide_buffer_t *dst, const void *src, size_t count) {
    tiramisu_cuda_memcpy_h2d((hipDeviceptr_t)(dst->device), src, count);
  }

  void htiramisu_cuda_memcpy_d2h(void *dst, halide_buffer_t *src, size_t count) {
    tiramisu_cuda_memcpy_d2h(dst, (hipDeviceptr_t)(src->device), count);
  }*/

}

#endif

#ifdef RUNTIME

extern "C" {

void tiramisu_init_stream_tracker(int max_streams) {
    assert(!st.initialized);
    st.streams = (hipStream_t *)malloc(sizeof(hipStream_t) * max_streams);
    st.init_streams = (bool *)calloc(max_streams, sizeof(bool)); // initialize to false
    st.initialized = true;
}

void tiramisu_cleanup_stream_tracker() {
    assert(st.initialized);
    free(st.streams);
    free(st.init_streams);
    st.initialized = false;
}

void tiramisu_cuda_malloc(void **device_ptr, size_t bytes) {
    assert(hipMalloc(device_ptr, bytes) == 0 && "tiramisu_cuda_malloc failed");
}

void tiramisu_cuda_memcpy_h2d(halide_buffer_t *dst, const void *src, size_t count, size_t dst_offset) {
    _tiramisu_cuda_memcpy_h2d(&(((float*)(dst->device))[dst_offset]), src, count);
}

void tiramisu_cuda_memcpy_d2h(void *dst, halide_buffer_t *src, size_t count, size_t src_offset) {
    _tiramisu_cuda_memcpy_d2h(dst, &(((float*)(src->device))[src_offset]), count);
}

void tiramisu_cuda_memcpy_h2d_async(halide_buffer_t *dst, const void *src, size_t count, size_t dst_offset, int stream_id,
                                    void *buff) {
    _tiramisu_cuda_memcpy_h2d_async(&(((float*)(dst->device))[dst_offset]), src, count, stream_id, buff);
}

void tiramisu_cuda_memcpy_d2h_async(void *dst, halide_buffer_t *src, size_t count, size_t src_offset,
                                    int stream_id, void *buff) {
    _tiramisu_cuda_memcpy_d2h_async(dst, &(((float*)(src->device))[src_offset]), count, stream_id, buff);
}

void tiramisu_cuda_free(void *device_ptr) {
    assert(hipFree(device_ptr) == 0 && "tiramisu_cuda_free failed");
}

void _tiramisu_cuda_memcpy_h2d(void *dst, const void *src, size_t count) {
    assert(hipMemcpy(dst, src, count, hipMemcpyHostToDevice) == 0 && "tiramisu_cuda_memcpy_h2d failed");
}

void _tiramisu_cuda_memcpy_h2h(void *dst, const void *src, size_t count) {
    assert(hipMemcpy(dst, src, count, hipMemcpyHostToHost) == 0 && "tiramisu_cuda_memcpy_h2h failed");
}

void _tiramisu_cuda_memcpy_d2h(void *dst, const void *src, size_t count) {
    assert(hipMemcpy(dst, src, count, hipMemcpyDeviceToHost) == 0 && "tiramisu_cuda_memcpy_d2h failed");
}

void _tiramisu_cuda_memcpy_d2d(void *dst, const void *src, size_t count) {
    assert(hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice) == 0 && "tiramisu_cuda_memcpy_d2d failed");
}

void _tiramisu_cuda_memcpy_h2d_async(void *dst, const void *src, size_t count, int stream_id, void *buff) {
    if (!st.init_streams[stream_id]) {
        st.streams[stream_id] = tiramisu_cuda_stream_create();
        st.init_streams[stream_id] = true;
    }
    hipStream_t stream = st.streams[stream_id];
    assert(hipMemcpyAsync(dst, src, count, hipMemcpyHostToDevice, stream) == 0 && "tiramisu_cuda_memcpy_h2d_async failed");
    // create the cuda event
    hipEvent_t event = tiramisu_cuda_event_create();
    tiramisu_cuda_event_record(event, stream);
    ((hipEvent_t*)buff)[0] = event;    
}

void _tiramisu_cuda_memcpy_d2h_async(void *dst, const void *src, size_t count, int stream_id, void *buff) {
    if (!st.init_streams[stream_id]) {
        st.streams[stream_id] = tiramisu_cuda_stream_create();
        st.init_streams[stream_id] = true;
    }
    hipStream_t stream = st.streams[stream_id];
    assert(hipMemcpyAsync(dst, src, count, hipMemcpyDeviceToHost, stream) == 0 && "tiramisu_cuda_memcpy_d2h_async failed");
    // create the cuda event
    hipEvent_t event = tiramisu_cuda_event_create();
    tiramisu_cuda_event_record(event, stream);
    ((hipEvent_t*)buff)[0] = event;
}

hipStream_t tiramisu_cuda_stream_create() {
    hipStream_t stream;
    assert(hipStreamCreate(&stream) == 0 && "tiramisu_cuda_stream_create failed");
    return stream;
}

void tiramisu_cuda_stream_destroy(hipStream_t stream) {
    assert(hipStreamDestroy(stream) == 0 && "tiramisu_cuda_stream_destroy failed");
}

void _tiramisu_cuda_stream_wait_event(hipStream_t stream, hipEvent_t event) {
    assert(hipStreamWaitEvent(stream, event, 0) == 0 && "tiramisu_cuda_stream_wait_event failed");
}

hipEvent_t tiramisu_cuda_event_create() {
    hipEvent_t event;
    assert(hipEventCreate(&event) == 0 && "tiramisu_cuda_event_create failed");
    return event;
}

void tiramisu_cuda_event_destroy(hipEvent_t event) {
    assert(hipEventDestroy(event) == 0 && "tiramisu_cuda_event_destroy failed");
}

void tiramisu_cuda_event_record(hipEvent_t event, hipStream_t stream) {
    assert(hipEventRecord(event, stream) == 0 && "tiramisu_cuda_event_record failed");
    assert(hipEventSynchronize(event) == 0 && "tiramisu_cuda_event_record synchronize failed");
}

void tiramisu_cuda_stream_wait_event(void *buff, int stream_id) {
  hipStream_t stream = st.streams[stream_id];
  hipEvent_t event = ((hipEvent_t)buff);
  _tiramisu_cuda_stream_wait_event(stream, event);
}

}
#endif