//
// Created by Jessica Ray on 12/6/17.
//

#include <cassert>
#include "tiramisu/tiramisu_cuda.h"

void tiramisu_cuda_malloc(void *device_ptr, size_t bytes) {
    assert(hipMalloc(&device_ptr, bytes) == 0 && "tiramisu_cuda_malloc failed");
}

void tiramisu_cuda_free(void *device_ptr) {
    assert(hipFree(device_ptr) == 0 && "tiramisu_cuda_free failed");
}

void tiramisu_cuda_memcpy_h2d(void *dst, const void *src, size_t count) {
    assert(hipMemcpy(dst, src, count, hipMemcpyHostToDevice) == 0 && "tiramisu_cuda_memcpy_h2d failed");
}

void tiramisu_cuda_memcpy_h2h(void *dst, const void *src, size_t count) {
    assert(hipMemcpy(dst, src, count, hipMemcpyHostToHost) == 0 && "tiramisu_cuda_memcpy_h2h failed");
}

void tiramisu_cuda_memcpy_d2h(void *dst, const void *src, size_t count) {
    assert(hipMemcpy(dst, src, count, hipMemcpyDeviceToHost) == 0 && "tiramisu_cuda_memcpy_d2h failed");
}

void tiramisu_cuda_memcpy_d2d(void *dst, const void *src, size_t count) {
    assert(hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice) == 0 && "tiramisu_cuda_memcpy_d2d failed");
}

void tiramisu_cuda_memcpy_h2d_async(void *dst, const void *src, size_t count, hipStream_t stream) {
    assert(false && "Not implemented yet");
    assert(hipMemcpyAsync(dst, src, count, hipMemcpyHostToDevice, stream) == 0 && "tiramisu_cuda_memcpy_h2d_async failed");
}

void tiramisu_cuda_memcpy_h2h_async(void *dst, const void *src, size_t count, hipStream_t stream) {
    assert(false && "Not implemented yet");
    assert(hipMemcpyAsync(dst, src, count, hipMemcpyHostToHost, stream) == 0 && "tiramisu_cuda_memcpy_h2h_async failed");
}

void tiramisu_cuda_memcpy_d2h_async(void *dst, const void *src, size_t count, hipStream_t stream) {
    assert(false && "Not implemented yet");
    assert(hipMemcpyAsync(dst, src, count, hipMemcpyDeviceToHost, stream) == 0 && "tiramisu_cuda_memcpy_d2h_async failed");
}

void tiramisu_cuda_memcpy_d2d_async(void *dst, const void *src, size_t count, hipStream_t stream) {
    assert(false && "Not implemented yet");
    assert(hipMemcpyAsync(dst, src, count, hipMemcpyDeviceToDevice, stream) == 0 && "tiramisu_cuda_memcpy_d2d_async failed");
}