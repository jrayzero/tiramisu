//
// Created by Jessica Ray on 12/6/17.
//

#include <cassert>
#include "tiramisu/tiramisu_cuda.h"
//#include "tiramisu/cuda_common.h"
#include <stdio.h>
#include "HalideRuntimeCuda.h"
#include "hip/hip_runtime.h"
//#include "device_buffer_utils.h"
//#include "device_interface.h"
//#include "printer.h"
//#include "mini_cuda.h"


#ifdef DRIVER
extern "C" {

  /*void tiramisu_cuda_malloc(hipDeviceptr_t *device_ptr, size_t bytes) {
  assert(hipMalloc(device_ptr, bytes) == 0 && "tiramisu_cuda_malloc failed");
}

void tiramisu_cuda_free(hipDeviceptr_t device_ptr) {
    assert(hipFree(device_ptr) == 0 && "tiramisu_cuda_free failed");
}

  void tiramisu_cuda_memcpy_h2d(hipDeviceptr_t dst, const void *src, size_t count) {
    assert(hipMemcpyHtoD(dst, src, count) == 0 && "tiramisu_cuda_memcpy_h2d failed");
  }

  void tiramisu_cuda_memcpy_d2h(void *dst, hipDeviceptr_t src, size_t count) {
    assert(hipMemcpyDtoH(dst, src, count) == 0 && "tiramisu_cuda_memcpy_d2h failed");
  }

  void htiramisu_cuda_malloc(halide_buffer_t *buff, size_t bytes) {
    hipDeviceptr_t p;
    tiramisu_cuda_malloc(&p, bytes);
    buff->device = p;
  }

  void htiramisu_cuda_free(halide_buffer_t *buff) {
    tiramisu_cuda_free((hipDeviceptr_t)(buff->device));
  }

  void htiramisu_cuda_memcpy_h2d(halide_buffer_t *dst, const void *src, size_t count) {
    tiramisu_cuda_memcpy_h2d((hipDeviceptr_t)(dst->device), src, count);
  }

  void htiramisu_cuda_memcpy_d2h(void *dst, halide_buffer_t *src, size_t count) {
    tiramisu_cuda_memcpy_d2h(dst, (hipDeviceptr_t)(src->device), count);
  }*/

}

#endif

#ifdef RUNTIME

extern "C" {

void *get_kernel_stream() {
    assert(st.init_kernel_stream);
    return *(st.kernel_stream);
}

hipStream_t tiramisu_get_kernel_stream() {
    assert(st.init_kernel_stream);
    return (hipStream_t)st.kernel_stream;
}

void tiramisu_init_stream_tracker(int max_streams, char *name) {
    assert(!st.initialized);
    st.comm_streams = (hipStream_t *)malloc(sizeof(hipStream_t) * max_streams);
    st.init_comm_streams = (bool *)calloc(max_streams, sizeof(bool)); // initialize to false
    st.initialized = true;
    //    if (!st.init_kernel_stream) {
    st.kernel_stream = (hipStream_t*)malloc(sizeof(hipStream_t));
    //      st.kernel_stream[0] = tiramisu_cuda_stream_create();      
    //      st.init_kernel_stream = true;
    //      fprintf(stderr, "query res %d\n", hipStreamQuery((ihipStream_t*)get_kernel_stream()));
    /*      if (err != 0) {
      fprintf(stderr, "%d\n", err);
    }
    st.kernel_stream = (hipStream_t)s;*/
    //      fprintf(stderr, "created kernel stream %p\n", st.kernel_stream);
    //    }
    st.nvvm_fname = name;
}

void tiramisu_cleanup_stream_tracker(int max_streams) {
    assert(st.initialized);
    for (int i = 0; i < max_streams; i++) {
        tiramisu_cuda_stream_destroy(st.comm_streams[i]);
    }
    tiramisu_cuda_stream_destroy(st.kernel_stream[0]);
    free(st.comm_streams);
    free(st.init_comm_streams);
    free(st.kernel_stream);
    st.initialized = false;
    st.init_kernel_stream = false;
    fprintf(stderr, "did stream tracker cleanup\n");
}

void tiramisu_cuda_malloc(void **device_ptr, size_t bytes) {
    assert(hipMalloc(device_ptr, bytes) == 0 && "tiramisu_cuda_malloc failed");
}

void tiramisu_cuda_memcpy_h2d(halide_buffer_t *dst, const void *src, size_t count, size_t dst_offset) {
    _tiramisu_cuda_memcpy_h2d(&(((float*)(dst->device))[dst_offset]), src, count);
}

void tiramisu_cuda_memcpy_d2h(void *dst, halide_buffer_t *src, size_t count, size_t src_offset) {
    _tiramisu_cuda_memcpy_d2h(dst, &(((float*)(src->device))[src_offset]), count);
}

void tiramisu_cuda_memcpy_h2d_async(halide_buffer_t *dst, const void *src, size_t count, size_t dst_offset, int stream_id,
                                    void *buff) {
    _tiramisu_cuda_memcpy_h2d_async(&(((float*)(dst->device))[dst_offset]), src, count, stream_id, buff);
}

void tiramisu_cuda_memcpy_d2h_async(void *dst, halide_buffer_t *src, size_t count, size_t src_offset,
                                    int stream_id, void *buff) {
    _tiramisu_cuda_memcpy_d2h_async(dst, &(((float*)(src->device))[src_offset]), count, stream_id, buff);
}

void tiramisu_cuda_free(void *device_ptr) {
    assert(hipFree(device_ptr) == 0 && "tiramisu_cuda_free failed");
}

void _tiramisu_cuda_memcpy_h2d(void *dst, const void *src, size_t count) {
    assert(hipMemcpy(dst, src, count, hipMemcpyHostToDevice) == 0 && "tiramisu_cuda_memcpy_h2d failed");
}

void _tiramisu_cuda_memcpy_h2h(void *dst, const void *src, size_t count) {
    assert(hipMemcpy(dst, src, count, hipMemcpyHostToHost) == 0 && "tiramisu_cuda_memcpy_h2h failed");
}

void _tiramisu_cuda_memcpy_d2h(void *dst, const void *src, size_t count) {
    assert(hipMemcpy(dst, src, count, hipMemcpyDeviceToHost) == 0 && "tiramisu_cuda_memcpy_d2h failed");
}

void _tiramisu_cuda_memcpy_d2d(void *dst, const void *src, size_t count) {
    assert(hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice) == 0 && "tiramisu_cuda_memcpy_d2d failed");
}

void _tiramisu_cuda_memcpy_h2d_async(void *dst, const void *src, size_t count, int stream_id, void *buff) {
    //  fprintf(stderr, "memcpy h2d async\n");
    if (!st.init_comm_streams[stream_id]) {
        st.comm_streams[stream_id] = tiramisu_cuda_stream_create();
        st.init_comm_streams[stream_id] = true;
    }
    hipStream_t stream = st.comm_streams[stream_id];
    assert(hipMemcpyAsync(dst, src, count, hipMemcpyHostToDevice, stream) == 0 && "tiramisu_cuda_memcpy_h2d_async failed");
    // create the cuda event
    hipEvent_t event = tiramisu_cuda_event_create();
    // You record the event of a particular stream. Later on you'd have another stream wait for this event. 
    // This gives you a way to synchronize across streams
    tiramisu_cuda_event_record(event, stream);//st.kernel_stream);//stream);
    ((hipEvent_t*)buff)[0] = event;
}

void _tiramisu_cuda_memcpy_d2h_async(void *dst, const void *src, size_t count, int stream_id, void *buff) {
    if (!st.init_comm_streams[stream_id]) {
        st.comm_streams[stream_id] = tiramisu_cuda_stream_create();
        st.init_comm_streams[stream_id] = true;
    }
    hipStream_t stream = st.comm_streams[stream_id];
    assert(hipMemcpyAsync(dst, src, count, hipMemcpyDeviceToHost, stream) == 0 && "tiramisu_cuda_memcpy_d2h_async failed");
    // create the cuda event
    hipEvent_t event = tiramisu_cuda_event_create();
    tiramisu_cuda_event_record(event, stream);//st.kernel_stream);//stream);
    ((hipEvent_t*)buff)[0] = event;
}

hipStream_t tiramisu_cuda_stream_create() {
    hipStream_t stream;
    assert(hipStreamCreate(&stream) == 0 && "tiramisu_cuda_stream_create failed");
    return stream;
}

void tiramisu_cuda_stream_destroy(hipStream_t stream) {
    assert(hipStreamDestroy(stream) == 0 && "tiramisu_cuda_stream_destroy failed");
}

void _tiramisu_cuda_stream_wait_event(hipStream_t stream, hipEvent_t event) {
    hipError_t res = hipStreamWaitEvent(stream, event, 0);
    if (res != 0) {
        fprintf(stderr, "%d, %p, %p\n", res, stream, event);
        assert(false);
    }
    //    assert(hipStreamWaitEvent(stream, event, 0) == 0 && "tiramisu_cuda_stream_wait_event failed");
}

hipEvent_t tiramisu_cuda_event_create() {
    hipEvent_t event;
    assert(hipEventCreate(&event) == 0 && "tiramisu_cuda_event_create failed");
    return event;
}

void tiramisu_cuda_event_destroy(hipEvent_t event) {
    assert(hipEventDestroy(event) == 0 && "tiramisu_cuda_event_destroy failed");
}

void tiramisu_cuda_event_record(hipEvent_t event, hipStream_t stream) {
    assert(hipEventRecord(event, stream) == 0 && "tiramisu_cuda_event_record failed");
}

/*  void tiramisu_cuda_stream_wait_event(void *buff, int stream_id) {
hipStream_t stream = st.comm_streams[stream_id];
hipEvent_t event = ((hipEvent_t)buff);
_tiramisu_cuda_stream_wait_event(stream, event);
}*/

void tiramisu_cuda_stream_wait_event(void *buff, int stream_id) {
    hipEvent_t event = ((hipEvent_t)buff);
    assert(st.kernel_stream[0] != NULL);
    _tiramisu_cuda_stream_wait_event(st.kernel_stream[0], event); // block the kernel stream on the communication event
}

int halide_launch_cuda_kernel(hipFunction_t f,int blocksX, int blocksY, int blocksZ,
                              int threadsX, int threadsY, int threadsZ,
                              int shared_mem_bytes, void **translated_args) {
    if (!st.init_kernel_stream) {
        hipStream_t stream = tiramisu_cuda_stream_create();//(hipStream_t)get_kernel_stream();//*((hipStream_t*)get_kernel_stream());
        st.kernel_stream[0] = stream;
        st.init_kernel_stream = true;
    }
    hipError_t err = hipModuleLaunchKernel(f,
                                  blocksX,  blocksY,  blocksZ,
                                  threadsX, threadsY, threadsZ,
                                  shared_mem_bytes,
                                  *(st.kernel_stream),
                                  translated_args,
                                  NULL);
    if (err != hipSuccess) {
        fprintf(stderr, "%d\n", err);
        assert(false && "hipModuleLaunchKernel failed");
    }
    return 0;

}


}
#endif